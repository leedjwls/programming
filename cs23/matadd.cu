
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS 10
#define N 100
#define A(i,j) A[i*N+j]
#define B(i,j) B[i*N+j]
#define C(i,j) C[i*N+j]

// CUDA kernel
__global__ void vecAdd(float *A, float *B, float *C)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   int j = blockDim.y * blockIdx.y + threadIdx.y; 

   C(i,j) = A(i,j) + B(i,j);
}

int main()
{
   float A[N][N], B[N][N], C[N][N], C2[N][N], *A_d, *B_d, *C_d;
   int i, j;

   dim3 dimBlock(THREADS, THREADS);
   dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);

   for (i=0; i<N; i++) {
      for (j=0; j<N; j++) {
         A[i][j] = i*2;
         B[i][j] = N-i;
         C2[i][j] = A[i][j] + B[i][j]; // C2 is used to check the results
      }
   }
   // Allocate array on device
   hipMalloc((void **)&A_d, sizeof(float)*N*N);
   hipMalloc((void **)&B_d, sizeof(float)*N*N);
   hipMalloc((void **)&C_d, sizeof(float)*N*N);

   // Copy array from host memory to device memory
   hipMemcpy(A_d, A, N*N*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(B_d, B, N*N*sizeof(float), hipMemcpyHostToDevice);

   vecAdd<<<dimGrid, dimBlock>>>(A_d, B_d, C_d);

   // Copy the result back
   hipMemcpy(C, C_d, N*N*sizeof(float), hipMemcpyDeviceToHost);

   // Check the results
   for (i=0; i<N; i++)
      for (j=0; j<N; j++)
         if (C[i][j] != C2[i][j]) printf("%d %f %f\n", i, C[i][j], C2[i][j]);

   /* Free memory on the device */
   hipFree(A_d);
   hipFree(B_d);
   hipFree(C_d);
}
