
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define N 10000
#define THREADS 100

__global__ void vecAdd(float *A, float *B)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   float X, Y;
   X = 1.23;
   Y = 2.34;

   B[i] = A[i]*X;
   B[i] += Y;
}

int main()
{
   float A[N], B[N], B2[N], X, Y, *A_d, *B_d;
   int i;

   dim3 dimBlock(THREADS);
   dim3 dimGrid(N/dimBlock.x);

   hipMalloc((void **)&A_d, sizeof(float)*N);
   hipMalloc((void **)&B_d, sizeof(float)*N);

   for (i=0; i<N; i++)
      A[i] = i*2;
  
   X = 1.23;
   Y = 2.34;
   for (i=0; i<N; i++) {
      B2[i] = A[i]*X + Y; // B2 is used for checking
   }

   hipMemcpy(A_d, A, N*sizeof(float), hipMemcpyHostToDevice);

   vecAdd<<<dimGrid, dimBlock>>>(A_d, B_d);

   hipMemcpy(B, B_d, N*sizeof(float), hipMemcpyDeviceToHost);

   // check the results
   for (i=0; i<N; i++)
      if (fabs(B[i]-B2[i]) > 0.001)
         printf("%d: %f %f\n",i, B[i], B2[i]);
}
