
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 24
#define THREADS 8

__global__ void reduce(float *A, float *results)
{
   __shared__ float sdata[THREADS];
   int i;
   unsigned int s;
   i = blockDim.x * blockIdx.x + threadIdx.x;
   sdata[threadIdx.x] = A[i];

   for(s = blockDim.x/2; s>0; s>>=1) {
      if(threadIdx.x < s)
         if(sdata[threadIdx.x] <= sdata[threadIdx.x + s])
            sdata[threadIdx.x] = sdata[threadIdx.x + s];
      __syncthreads();
   }
   if(threadIdx.x == 0) results[blockIdx.x] = sdata[0];
}

int main()
{
   float A[N], *A_d, *results, *results_d, result;
   int i;

   dim3 dimBlock(THREADS);
   dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);
   hipSetDevice(0);

   for (i=0; i<N; i++)
      A[i] = N-i;
   A[3] = 2*N; A[N-3] = -N;

   hipMalloc((void **) &A_d, sizeof(float)*N);
   hipMemcpy(A_d, A, sizeof(float)*N, hipMemcpyHostToDevice);

   hipMalloc((void **) &results_d, dimGrid.x*sizeof(float));

   reduce<<<dimGrid, dimBlock>>>(A_d, results_d);

   // copy values from GPU memory to CPU memory
   results = (float*)malloc(dimGrid.x*sizeof(float));
   hipMemcpy(results, results_d, dimGrid.x*sizeof(float), hipMemcpyDeviceToHost);

   // add the computed value
   result = 999999.0;
   for (i=0; i<dimGrid.x; i++)
      if(result < results[i]) result = results[i];

   printf("%f\n", result);

   hipFree(A_d);
   hipFree(results_d);
}
