#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 128000
#define THREADS 128

__global__ void reduce(float *sum, float dx)
{
   __shared__ float sdata[THREADS];
   unsigned int s;
   float x;
   int i;
   i = blockDim.x * blockIdx.x + threadIdx.x;
   x = dx * i;
   sdata[threadIdx.x] = (x/((x*x+2)*(x*x+2)*(x*x+2)))*dx;

   for(s = blockDim.x/2; s>0; s>>=1) {
      if(threadIdx.x < s)
         sdata[threadIdx.x] += sdata[threadIdx.x + s];
      __syncthreads();
   }
   if(threadIdx.x == 0) sum[blockIdx.x] = sdata[0];
}

main(int argc, char *argv[]) {
   float *area, *area_d, dx, a, b, elapsed_time;
   int i;
   b = 2.0;
   a = 0.0;

   dim3 dimBlock(THREADS);
   dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);

   dx = (b-a) / N;
   
   hipSetDevice(0);
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);

   hipMalloc((void **) &area_d, dimGrid.x*sizeof(float));

   reduce<<<dimGrid, dimBlock>>>(area_d, dx);

   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsed_time, start, stop);

   area = (float*)malloc(dimGrid.x*sizeof(float));
   hipMemcpy(area, area_d, dimGrid.x*sizeof(float), hipMemcpyDeviceToHost);

   for(i=0; i<N/THREADS; i++) {
      area[0] += area[i];
   }
   
   printf("elapsed time: %f ms\n", elapsed_time);
   printf("result: %f\n", area[0]);

   hipFree(area_d);
}
